/*
 
Copyright (c) 2004, Cornell University
All rights reserved.
 
Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
 
   - Redistributions of source code must retain the above copyright notice,
       this list of conditions and the following disclaimer.
   - Redistributions in binary form must reproduce the above copyright
       notice, this list of conditions and the following disclaimer in the
       documentation and/or other materials provided with the distribution.
   - Neither the name of Cornell University nor the names of its
       contributors may be used to endorse or promote products derived from
       this software without specific prior written permission.
 
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
 
*/


/////////////////////////////////////////////////////////////////////
///
/// Bitmap8.cpp
///
/////////////////////////////////////////////////////////////////////

#include <stdio.h>
#include "Bitmap8.h"
#include "Bitmaps_cuda.h"

const int Bitmap8::_lookupTableSize = 0x10000;

int Bitmap8::_countOr = 0;
int Bitmap8::_countAnd = 0;
int Bitmap8::_countCount = 0;
int Bitmap8::_countCreateSBitmap = 0;
int* Bitmap8::_countLookupTable = 0;
int* Bitmap8::_sBitmapLookupTable = 0;
int* Bitmap8::_cBitmapLookupTable = 0;

#define DEBUG_BITMAP8COUNT

/////////////////////////////////////////////////////////////////////
/// Initialize the counting tables
/////////////////////////////////////////////////////////////////////
void Bitmap8::Init()
{
    int i, s;
    int i1, i2, a1, a2;

    // ------ initialize _countLookupTable -----------------------------
    _countLookupTable = new int[_lookupTableSize];
    memset(_countLookupTable, 0, SIZE_INT*_lookupTableSize);
    for (i = 1; i < _lookupTableSize; i++)
    {
        if (i & 0x00ff)
            _countLookupTable[i]++;
        if (i & 0xff00)
            _countLookupTable[i]++;
        assert(1 == _countLookupTable[i] || 2 == _countLookupTable[i]);
    }

    // ------ initialize _sBitmapLookupTable -----------------------------
    // note: for a customer (4 bits), set the first bit
    // (after the first bit with a one) to one
    // recall that a SHORT is 16 bits, so we need to
    // change bits for 4 customers

    int Bit8SBitmapLookUp[256];
    Bit8SBitmapLookUp[0] = 0;
    Bit8SBitmapLookUp[1] = 0;

    int curValue = 0;
    int curIndex = 1;
    for (i = 2; i < 256; i++)
    {
        if (i % curIndex == 0)
        {
            curValue = curValue + curIndex;
            curIndex *= 2;
        }
        Bit8SBitmapLookUp[i] = curValue;
    }

    _sBitmapLookupTable = new int[_lookupTableSize];
    memset(_sBitmapLookupTable, 0, SIZE_INT*_lookupTableSize);

    s = 0;  // index into the sBitmapLookupTable
    for (i1 = 0; i1 < 256; i1++)
    {
        // first customer
        a1 = Bit8SBitmapLookUp[i1] << 8;

        for (i2 = 0; i2 < 256; i2++)
        {
            // second customer
            a2 = Bit8SBitmapLookUp[i2];

            // now actually set the sBitmapLookupTable value
            _sBitmapLookupTable[s] = a1 | a2;
            s++;

        } // for i2
    } // for i1

	// GPU Variables
	int _sBitmapLookupTable8_device[_lookupTableSize];
	int _countLookupTable8_device[_lookupTableSize];

	// Memory alloc
	hipMalloc((void**)&_sBitmapLookupTable8_device, SIZE_INT*_lookupTableSize);
	hipMalloc((void**)&_countLookupTable8_device, SIZE_INT*_lookupTableSize);

	// copy in memory
	hipMemcpyToSymbol(HIP_SYMBOL("_sBitmapLookupTable8_device"), &_sBitmapLookupTable[0], SIZE_INT*_lookupTableSize);
	hipMemcpyToSymbol(HIP_SYMBOL("_countLookupTable8_device"), &_countLookupTable[0], SIZE_INT*_lookupTableSize);
}

/////////////////////////////////////////////////////////////////////
/// deallocate counting tables
/////////////////////////////////////////////////////////////////////
void Bitmap8::Destroy()
{
    if (_countLookupTable != 0)
        delete [] _countLookupTable;

    if (_sBitmapLookupTable != 0)
        delete [] _sBitmapLookupTable;
}

/////////////////////////////////////////////////////////////////////
/// Bitwise OR 2 bitmap8s and store the result
///
/// @param b1                the first Bitmap8
/// @param b2                the second Bitmap8
/////////////////////////////////////////////////////////////////////
void Bitmap8::Or(const Bitmap8 &b1, const Bitmap8 &b2)
{
#ifdef DEBUG_BITMAP8COUNT
    _countOr++;
#endif

    // OR each int bitwise
    for (int i = 0; i < b1._memSizeInt; i++)
        _memory[i] = b1._memory[i] | b2._memory[i];
}

/////////////////////////////////////////////////////////////////////
/// Bitwise AND 2 bitmap8s and store the result
///
/// @param b1                the first Bitmap8
/// @param b2                the second Bitmap8
/////////////////////////////////////////////////////////////////////
void Bitmap8::And(const Bitmap8 &b1, const Bitmap8 &b2)
{

#ifdef DEBUG_BITMAP8COUNT
    _countAnd++;
#endif

	// For Candidate wise or Transaction wise technique
	// delete the comment in Bitmaps_cuda.cu

	//declare the events
	hipEvent_t start;
	hipEvent_t stop;
	float kernel_time;

	//create events before you use them
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// device arrays
	unsigned int* _memory_device;
	unsigned int* b1_memory;
	unsigned int* b2_memory;
	
	int b1_size = b1.getIntSize();

	// allocate memory on GPU
	hipMalloc((void **)&b1_memory,  _memSizeInt * SIZE_UINT);
	hipMalloc((void **)&b2_memory,  _memSizeInt * SIZE_UINT);
	hipMalloc((void **)&_memory_device,  _memSizeInt * SIZE_UINT);

	// copy values on GPU
	hipMemcpy(b1_memory, b1._memory, _memSizeInt * SIZE_UINT, hipMemcpyHostToDevice );
	hipMemcpy(b2_memory, b2._memory, _memSizeInt * SIZE_UINT, hipMemcpyHostToDevice );
	hipMemcpy(_memory_device, _memory, _memSizeInt * SIZE_UINT, hipMemcpyHostToDevice );

	// need to verify it
	dim3 dimBlock(256, 1, 1);
	dim3 dimGrid(256, 1);

	//put events and kernel launches in the stream/queue
	hipEventRecord(start,0);
	// Launch kernel 
	AndBitwiseOperation<<<dimGrid, dimBlock>>>(_memory_device, b1_size, b1_memory, b2_memory);
	hipEventRecord(stop,0);

	//wait until the stop event is recorded
	hipEventSynchronize(stop);

	//and get the elapsed time
	hipEventElapsedTime(&kernel_time,start,stop);
	
	/* need to divide the elapsed time by number of blok and number of threads
	   to get the correct time of a single "function call" => it is divided by 1000
	   for getting a better visualization of the time
	 */
	// kernel_time = (kernel_time/(n_blocks * n_threads))/1000;
	// printf("Bitmap4 time taken: %n\n", kernel_time);

	//cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Wait until the kernel has finished
	hipDeviceSynchronize();

	// return values
	hipMemcpy(_memory, _memory_device, _memSizeInt * SIZE_UINT, hipMemcpyDeviceToHost );

	// Free Memory
	hipFree(b1_memory);
	hipFree(b2_memory);
	hipFree(_memory_device);
}

/////////////////////////////////////////////////////////////////////
/// find the support of this bitmap in *number of customers*
///
/// @return the number of customers that have some bit set among their 8 bits
/////////////////////////////////////////////////////////////////////
int Bitmap8::Count()
{

#ifdef DEBUG_BITMAP8COUNT
    _countCount++;
#endif

	int support = 0;

    // we walk the memory in terms of shorts
    // ms: pointer to memory as a pointer to shorts
    // ss: size of the memory in short
    unsigned short* ms = reinterpret_cast<unsigned short*>(_memory);
    
    //declare the events
	hipEvent_t start;
	hipEvent_t stop;
	float kernel_time;

	//create events before you use them
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// GPU Variable
	unsigned short* ms_device;
	int* support_device;
	int temp = _memSizeShort;
	int* _memSizeShort_device;

	// Memory alloc
	hipMalloc((void**)&ms_device, sizeof(unsigned short*));
	hipMalloc((void**)&support_device, sizeof(int*));
	hipMalloc((void**)&_memSizeShort_device, sizeof(int*));

	// Copy in GPU
	hipMemcpy(ms_device, ms, sizeof(unsigned short*), hipMemcpyHostToDevice);
	hipMemcpy(&support_device[0], &support, sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(_memSizeShort_device, &temp, sizeof(int*), hipMemcpyHostToDevice);

	// kernel parameters
	dim3 dimBlock(16, 1);
	dim3 dimGrid(1, 1, 1);

	//put events and kernel launches in the stream/queue
	hipEventRecord(start,0);
	Count8_global<<<dimGrid, dimBlock>>>(ms_device, support_device, _memSizeShort_device);
	hipEventRecord(stop,0);

	//wait until the stop event is recorded
	hipEventSynchronize(stop);

	//and get the elapsed time
	hipEventElapsedTime(&kernel_time,start,stop);
	
	/* need to divide the elapsed time by number of blok and number of threads
	   to get the correct time of a single "function call" => it is divided by 1000
	   for getting a better visualization of the time
	 */
	// kernel_time = (kernel_time/(n_blocks * n_threads))/1000;
	// printf("Bitmap4 time taken: %n\n", kernel_time);

	//cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	// Wait until the kernel has finished
	hipDeviceSynchronize();

	// Copy back
	hipMemcpy(ms, ms_device, sizeof(unsigned short*), hipMemcpyDeviceToHost);
	hipMemcpy(&support, &support_device[0], sizeof(int*), hipMemcpyDeviceToHost);

	// free memory
	hipFree(ms_device);
	hipFree(support_device);
	
	return support;
}

/////////////////////////////////////////////////////////////////////
/// Create a s-bitmap from an i-bitmap
/// <p>
/// Idea  : Again, we go thru each element of _memory. For each element,
///     if it is greater than 0, we look up the transformation table
///     (postProcessTable) to find the corresponding value for the s-bitmap,
///     and set the remaining SHORTs of the current custom to USHRT_MAX
///     (i.e. all 1's).
/// <p>
/// Note  : For example, if the bitmap is
///     [0001 | 1100 | 0011 | 1111 | 0000 | 0000] and
///     [00001111 | 00011111 | 11111111]. Refer to the paper for details.
///
/// @param iBitmap           the bitmap8 from which we create s-bitmap
/////////////////////////////////////////////////////////////////////
void Bitmap8::CreateSBitmap(const Bitmap8 &iBitmap)
{

#ifdef DEBUG_BITMAP8COUNT
    _countCreateSBitmap++;
#endif

    assert(_memory);
    assert(_memSizeShort == iBitmap._memSizeShort);

    unsigned short* ms = reinterpret_cast<unsigned short*>(_memory);
    const unsigned short* msib = reinterpret_cast<const unsigned short*> (iBitmap._memory);
    
    //declare the events
	hipEvent_t start;
	hipEvent_t stop;
	float kernel_time;

	//create events before you use them
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// GPU Variables
	unsigned short* ms_device;
	unsigned short* msib_device;
	int _memSizeShort = iBitmap._memSizeShort;
	int* _memSizeShort_device; 

	// Memory alloc
	hipMalloc((void**)&ms_device, sizeof(unsigned short*));
	hipMalloc((void**)&msib_device, sizeof(const unsigned short*));
	hipMalloc((void**)&_memSizeShort_device, sizeof(int*));

	// copy in GPU
	hipMemcpy(ms_device, ms, sizeof(unsigned short*), hipMemcpyHostToDevice);
	hipMemcpy(msib_device, msib, sizeof(const unsigned short*), hipMemcpyHostToDevice);
	hipMemcpy(_memSizeShort_device, &_memSizeShort, sizeof(int*), hipMemcpyHostToDevice);

	// kernel parameters
	dim3 dimBlock(256, 1);
	dim3 dimGrid(256, 1, 1);

	//put events and kernel launches in the stream/queue
	hipEventRecord(start,0);
	CreateSBitmap8<<<dimGrid, dimBlock>>>(ms_device, msib_device, _memSizeShort_device);
	hipEventRecord(stop,0);

	//wait until the stop event is recorded
	hipEventSynchronize(stop);

	//and get the elapsed time
	hipEventElapsedTime(&kernel_time,start,stop);
	
	/* need to divide the elapsed time by number of blok and number of threads
	   to get the correct time of a single "function call" => it is divided by 1000
	   for getting a better visualization of the time
	 */
	// kernel_time = (kernel_time/(n_blocks * n_threads))/1000;
	// printf("Bitmap4 time taken: %n\n", kernel_time);

	//cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	// Wait until the kernel has finished
	hipDeviceSynchronize();
	
	// copy back
	hipMemcpy(ms, ms_device, sizeof(unsigned short*), hipMemcpyDeviceToHost);

	// free memory
	hipFree(ms_device);
	hipFree(msib_device);
}

