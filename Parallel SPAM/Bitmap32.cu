/*
 
Copyright (c) 2004, Cornell University
All rights reserved.
 
Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
 
   - Redistributions of source code must retain the above copyright notice,
       this list of conditions and the following disclaimer.
   - Redistributions in binary form must reproduce the above copyright
       notice, this list of conditions and the following disclaimer in the
       documentation and/or other materials provided with the distribution.
   - Neither the name of Cornell University nor the names of its
       contributors may be used to endorse or promote products derived from
       this software without specific prior written permission.
 
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
 
*/


/////////////////////////////////////////////////////////////////////
///
/// Bitmap32.cpp
///
/////////////////////////////////////////////////////////////////////

#include <stdio.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include "Bitmaps_cuda.h"
#include "Bitmap32.h"

const int Bitmap32::_lookupTableSize = 0x10000;

int Bitmap32::_countOr = 0;
int Bitmap32::_countAnd = 0;
int Bitmap32::_countCount = 0;
int Bitmap32::_countCreateSBitmap = 0;
int* Bitmap32::_sBitmapLookupTable = 0;
int* Bitmap32::_cBitmapLookupTable = 0;

#define DEBUG_BITMAP32COUNT

/////////////////////////////////////////////////////////////////////
/// Initialize the counting tables
/////////////////////////////////////////////////////////////////////
void Bitmap32::Init()
{

    int i;

    // ------ initialize sBitmapLookupTable -----------------------------
    _sBitmapLookupTable = new int[_lookupTableSize];
    memset(_sBitmapLookupTable, 0, SIZE_INT*_lookupTableSize);

    _sBitmapLookupTable[0] = 0;
    _sBitmapLookupTable[1] = 0;

    int curValue = 0;
    int curIndex = 1;
    for (i = 2; i < _lookupTableSize; i++)
    {
        if (i % curIndex == 0)
        {
            curValue = curValue + curIndex;
            curIndex *= 2;
        }
        _sBitmapLookupTable[i] = curValue;
    }

	// GPU Variables
	int _sBitmapLookupTable32_device[_lookupTableSize];

	// Memory alloc
	hipMalloc((void**)&_sBitmapLookupTable32_device, SIZE_INT*_lookupTableSize);

	// copy in memory
	hipMemcpyToSymbol(HIP_SYMBOL("_sBitmapLookupTable32_device"), &_sBitmapLookupTable[0], SIZE_INT*_lookupTableSize);
}

/////////////////////////////////////////////////////////////////////
/// deallocate counting tables
/////////////////////////////////////////////////////////////////////
void Bitmap32::Destroy()
{
    if (_sBitmapLookupTable != 0)
        delete [] _sBitmapLookupTable;
}

/////////////////////////////////////////////////////////////////////
/// Bitwise OR 2 bitmap32s and store the result
///
/// @param b1                the first Bitmap32
/// @param b2                the second Bitmap32
/////////////////////////////////////////////////////////////////////
void Bitmap32::Or(const Bitmap32 &b1, const Bitmap32 &b2)
{

#ifdef DEBUG_BITMAP32COUNT
    _countOr++;
#endif

    // OR each int bitwise
    for (int i = 0; i < b1._memSizeInt; i++)
        _memory[i] = b1._memory[i] | b2._memory[i];
}


/////////////////////////////////////////////////////////////////////
/// Bitwise AND 2 bitmaps32 and store the result
///
/// @param b1                the first Bitmap32
/// @param b2                the second Bitmap32
/////////////////////////////////////////////////////////////////////
void Bitmap32::And(const Bitmap32 &b1, const Bitmap32 &b2)
{

#ifdef DEBUG_BITMAP32COUNT
    _countAnd++;
#endif

	// For Candidate wise or Transaction wise technique
	// delete the comment in Bitmaps_cuda.cu

	//declare the events
	hipEvent_t start;
	hipEvent_t stop;
	float kernel_time;

	//create events before you use them
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// device arrays
	unsigned int* _memory_device;
	unsigned int* b1_memory;
	unsigned int* b2_memory;
	
	int b1_size = b1.getIntSize();

	// allocate memory on GPU
	hipMalloc((void **)&b1_memory,  _memSizeInt * SIZE_UINT);
	hipMalloc((void **)&b2_memory,  _memSizeInt * SIZE_UINT);
	hipMalloc((void **)&_memory_device,  _memSizeInt * SIZE_UINT);

	// copy values on GPU
	hipMemcpy(b1_memory, b1._memory, _memSizeInt * SIZE_UINT, hipMemcpyHostToDevice );
	hipMemcpy(b2_memory, b2._memory, _memSizeInt * SIZE_UINT, hipMemcpyHostToDevice );
	hipMemcpy(_memory_device, _memory, _memSizeInt * SIZE_UINT, hipMemcpyHostToDevice );

	// need to verify it
	dim3 dimBlock(256, 1, 1);
	dim3 dimGrid(256, 1);

	//put events and kernel launches in the stream/queue
	hipEventRecord(start,0);
	// Launch kernel
	AndBitwiseOperation<<<dimGrid, dimBlock>>>(_memory_device, b1_size, b1_memory, b2_memory);
	hipEventRecord(stop,0);

	//wait until the stop event is recorded
	hipEventSynchronize(stop);

	//and get the elapsed time
	hipEventElapsedTime(&kernel_time,start,stop);
	
	/* need to divide the elapsed time by number of blok and number of threads
	   to get the correct time of a single "function call" => it is divided by 1000
	   for getting a better visualization of the time
	 */
	// kernel_time = (kernel_time/(n_blocks * n_threads))/1000;
	// printf("Bitmap4 time taken: %n\n", kernel_time);

	//cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Wait until the kernel has finished
	hipDeviceSynchronize();

	// return values
	hipMemcpy(_memory, _memory_device, _memSizeInt * SIZE_UINT, hipMemcpyDeviceToHost );

	// Free Memory
	hipFree(b1_memory);
	hipFree(b2_memory);
	hipFree(_memory_device);
}


/////////////////////////////////////////////////////////////////////
/// find the support of this bitmap in *number of customers*
///
/// @return the number of customers that have some bit set among their 32 bits
/////////////////////////////////////////////////////////////////////
int Bitmap32::Count()
{

#ifdef DEBUG_BITMAP32COUNT
    _countCount++;
#endif

	int support = 0;
	
	//declare the events
	hipEvent_t start;
	hipEvent_t stop;
	float kernel_time;

	//create events before you use them
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// GPU Variable
	unsigned short* ms_device;
	int* support_device;
	int temp = _memSizeShort;
	int* _memSizeShort_device;

	// Memory alloc
	hipMalloc((void**)&ms_device, sizeof(unsigned short*));
	hipMalloc((void**)&support_device, sizeof(int*));
	hipMalloc((void**)&_memSizeShort_device, sizeof(int*));

	// Copy in GPU
	hipMemcpy(ms_device, _memory, sizeof(unsigned short*), hipMemcpyHostToDevice);
	hipMemcpy(support_device, &support, sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(&_memSizeShort_device[0], &temp, sizeof(int*), hipMemcpyHostToDevice);

	// kernel parameters - one warp
	dim3 dimBlock(1, 1);
	dim3 dimGrid(1, 1, 1);

	//put events and kernel launches in the stream/queue
	hipEventRecord(start,0);
	Count32_global<<<dimGrid, dimBlock>>>(ms_device, support_device, _memSizeShort_device);
	hipEventRecord(stop,0);

	//wait until the stop event is recorded
	hipEventSynchronize(stop);

	//and get the elapsed time
	hipEventElapsedTime(&kernel_time,start,stop);
	
	/* need to divide the elapsed time by number of blok and number of threads
	   to get the correct time of a single "function call" => it is divided by 1000
	   for getting a better visualization of the time
	 */
	// kernel_time = (kernel_time/(n_blocks * n_threads))/1000;
	// printf("Bitmap4 time taken: %n\n", kernel_time);

	//cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Wait until the kernel has finished
	hipDeviceSynchronize();

	// Copy back
	hipMemcpy(_memory, ms_device, sizeof(unsigned short*), hipMemcpyDeviceToHost);
	hipMemcpy(&support, support_device, sizeof(int*), hipMemcpyDeviceToHost);

	// free memory
	hipFree(ms_device);
	hipFree(support_device);

	return support;
}

/////////////////////////////////////////////////////////////////////
/// Create a s-bitmap from an i-bitmap
/// <p>
/// Idea  : Again, we go thru each element of _memory. For each element,
///     if it is greater than 0, we look up the transformation table
///     (postProcessTable) to find the corresponding value for the s-bitmap,
///     and set the remaining SHORTs of the current custom to USHRT_MAX
///     (i.e. all 1's).
/// <p>
/// Note  : For example, if the bitmap is
///     [0001 | 1100 | 0011 | 1111 | 0000 | 0000] and
///     [00001111 | 00011111 | 11111111]. Refer to the paper for details.
///
/// @param iBitmap           the bitmap32 from which we create s-bitmap
/////////////////////////////////////////////////////////////////////
void Bitmap32::CreateSBitmap(const Bitmap32 &iBitmap)
{

#ifdef DEBUG_BITMAP32COUNT
    _countCreateSBitmap++;
#endif

    assert(_memory);
    assert(_memSizeInt == iBitmap._memSizeInt);
    
	//declare the events
	hipEvent_t start;
	hipEvent_t stop;
	float kernel_time;

	//create events before you use them
	hipEventCreate(&start);
	hipEventCreate(&stop);    

    unsigned short* ms = reinterpret_cast<unsigned short*>(_memory);
    const unsigned short* msib = reinterpret_cast<const unsigned short*> (iBitmap._memory);

	// GPU Variables
	unsigned short* ms_device;
	unsigned short* msib_device;
	int _memSizeShort = iBitmap._memSizeShort;
	int* _memSizeShort_device; 

	// Memory alloc
	hipMalloc((void**)&ms_device, sizeof(unsigned short*));
	hipMalloc((void**)&msib_device, sizeof(const unsigned short*));
	hipMalloc((void**)&_memSizeShort_device, sizeof(int*));

	// copy in GPU
	hipMemcpy(ms_device, ms, sizeof(unsigned short*), hipMemcpyHostToDevice);
	hipMemcpy(msib_device, msib, sizeof(const unsigned short*), hipMemcpyHostToDevice);
	hipMemcpy(_memSizeShort_device, &_memSizeShort, sizeof(int*), hipMemcpyHostToDevice);

	// kernel parameters
	dim3 dimBlock(1, 1);
	dim3 dimGrid(1, 1, 1);

	//put events and kernel launches in the stream/queue
	hipEventRecord(start,0);
	CreateSBitmap32<<<dimGrid, dimBlock>>>(ms_device, msib_device, _memSizeShort_device);
	hipEventRecord(stop,0);

	//wait until the stop event is recorded
	hipEventSynchronize(stop);

	//and get the elapsed time
	hipEventElapsedTime(&kernel_time,start,stop);
	
	/* need to divide the elapsed time by number of blok and number of threads
	   to get the correct time of a single "function call" => it is divided by 1000
	   for getting a better visualization of the time
	 */
	// kernel_time = (kernel_time/(n_blocks * n_threads))/1000;
	// printf("Bitmap4 time taken: %n\n", kernel_time);

	//cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	// Wait until the kernel has finished
	hipDeviceSynchronize();
	
	// copy back
	hipMemcpy(ms, ms_device, sizeof(unsigned short*), hipMemcpyDeviceToHost);

	// free memory
	hipFree(ms_device);
	hipFree(msib_device);
}
